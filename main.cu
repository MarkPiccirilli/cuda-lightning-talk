
#include <hip/hip_runtime.h>
#include <omp.h>
#include <cstdlib>
#include <iostream>
#include <math.h>

#ifndef ARRAYSIZE
#define ARRAYSIZE 1000000
#endif

//#ifndef THREADARRAY
//#define THREADARRAY (int[]){1, 2, 4, 6, 8, 12, 16}
//#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE 256
#endif

#ifndef NUMBLOCKS
#define NUMBLOCKS 0
#endif

using std::cout;
using std::cerr;

__global__
void cudaMultiply(int *array1, int *array2, int *array3, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < size; i += stride)
        array3[i] = array1[i] * array2[i];
}

void printArray(int *array, int arraySize) {
    for(int i = 0; i < arraySize; i++) {
        cout << array[i] << " ";
    }
    cout << '\n';
}

int main(int argc, char **argv) {

    #ifndef _OPENMP
    cerr << "No OpenMP support";
    return 1;
    #endif

    int arraySize = ARRAYSIZE;
    int threadArray[] = {1, 2, 4, 6, 8, 12, 16};
    int threadArraySize = sizeof(threadArray)/sizeof(threadArray[0]);

    int* hostArray1 = new int[arraySize];
    int* hostArray2 = new int[arraySize];
    int* hostArray3 = new int[arraySize];

    for(int i = 0; i < arraySize; i++) {
        hostArray1[i] = rand() % 100;
        hostArray2[i] = rand() % 100;
    }

    for(int i = 0; i < threadArraySize; i++) {
        omp_set_num_threads(threadArray[i]);
        double timeStart = omp_get_wtime();

        #pragma omp parallel for
        for(int j = 0; j < arraySize; j++) {
            hostArray3[j] = hostArray1[j] * hostArray2[j];
        }

        double timeComplete = omp_get_wtime();
        double totalTime = timeComplete - timeStart;
        cout << "Time to complete operation with " << threadArray[i] << " threads(ms): " << totalTime * 1000 << '\n';
    }

    int *deviceArray1, *deviceArray2, *deviceArray3;
    hipMallocManaged(&deviceArray1, arraySize * sizeof(int));
    hipMallocManaged(&deviceArray2, arraySize * sizeof(int));
    hipMallocManaged(&deviceArray3, arraySize * sizeof(int));

    hipMemcpy(deviceArray1, hostArray1, arraySize * sizeof(int), hipMemcpyHostToDevice);

    //allocate CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int blockSize = BLOCKSIZE;
    int numBlocks = NUMBLOCKS ? (arraySize + blockSize - 1) / blockSize : NUMBLOCKS;

    hipEventRecord(start, NULL);
    cudaMultiply<<<numBlocks, blockSize>>>(deviceArray1, deviceArray2, deviceArray3, arraySize);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);

//    cudaDeviceSynchronize();

    float totalTimeGPU = 0.0f;
    hipEventElapsedTime(&totalTimeGPU, start, stop);
    cout << "Time to complete operation with the GPU using blocksize: " << blockSize << " and numBlocks: " << numBlocks << " was(ms): " << totalTimeGPU << '\n';

    // Free memory
    delete [] hostArray1;
    delete [] hostArray2;
    delete [] hostArray3;
    hipFree(deviceArray1);
    hipFree(deviceArray2);
    hipFree(deviceArray3);

    return 0;
}
