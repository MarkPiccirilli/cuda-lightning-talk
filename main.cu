
#include <hip/hip_runtime.h>
#include <omp.h>
#include <cstdlib>
#include <iostream>
#include <math.h>

#ifndef ARRAYSIZE
#define ARRAYSIZE 1000000000
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE 256
#endif

//#ifndef NUMBLOCKS
//#define NUMBLOCKS NULL
//#endif

#ifndef NUMTRIES
#define NUMTRIES 50
#endif

using std::cout;
using std::cerr;
using std:: endl;

__global__
void cudaMultiply(long long *array1, long long *array2, long long *array3, long long size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < size; i += stride)
        array3[i] = array1[i] * array2[i];
}

int main(int argc, char **argv) {

    #ifndef _OPENMP
    cerr << "No OpenMP support";
    return 1;
    #endif

    long long arraySize = ARRAYSIZE;
    cout << "ArraySize: " << arraySize << endl;
    cout << "NumTries: " << NUMTRIES << endl;
    int threadArray[] = {1, 2, 4, 6, 8, 12, 16};
    int threadArraySize = sizeof(threadArray)/sizeof(threadArray[0]);

    long long* hostArray1 = new long long[arraySize];
    long long* hostArray2 = new long long[arraySize];
    long long* hostArray3 = new long long[arraySize];

    for(int i = 0; i < arraySize; i++) {
        hostArray1[i] = rand() % 100;
        hostArray2[i] = rand() % 100;
    }

    for(int i = 0; i < threadArraySize; i++) {
        omp_set_num_threads(threadArray[i]);

        double totalTime = 0;
        for(int j = 0; j < NUMTRIES; j++) {
            double timeStart = omp_get_wtime();
            #pragma omp parallel for
            for(int k = 0; k < arraySize; k++) {
                hostArray3[k] = hostArray1[k] * hostArray2[k];
            }
            double timeComplete = omp_get_wtime();
            totalTime = totalTime + timeComplete - timeStart;
        }
        double averageTime = totalTime / NUMTRIES;

        cout << "Time to complete operation with " << threadArray[i] << " threads(ms): " << averageTime * 1000 << endl;
    }

    long long *deviceArray1, *deviceArray2, *deviceArray3;
    hipMallocManaged(&deviceArray1, arraySize * sizeof(long long));
    hipMallocManaged(&deviceArray2, arraySize * sizeof(long long));
    hipMallocManaged(&deviceArray3, arraySize * sizeof(long long));

    hipMemcpy(deviceArray1, hostArray1, arraySize * sizeof(long long), hipMemcpyHostToDevice);

    //allocate CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int blockSize = BLOCKSIZE;
    //int numBlocks = NUMBLOCKS ? NUMBLOCKS : (arraySize + blockSize - 1) / blockSize;
    int numBlocks = (arraySize + blockSize - 1) / blockSize;
	
    double totalTimeGPU = 0.0;
    for(int i = 0; i < NUMTRIES; i++) {
        hipEventRecord(start, NULL);
        cudaMultiply<<<numBlocks, blockSize>>>(deviceArray1, deviceArray2, deviceArray3, arraySize);
        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop);

        float runTimeGPU = 0.0f;
        hipEventElapsedTime(&runTimeGPU, start, stop);

        totalTimeGPU = totalTimeGPU + runTimeGPU;
    }

    double averageTimeGPU = totalTimeGPU / NUMTRIES;

    cout << "Time to complete operation with the GPU using blocksize: " << blockSize << " and numBlocks: " << numBlocks << " was(ms): " << averageTimeGPU << endl;

    // Free memory
    delete [] hostArray1;
    delete [] hostArray2;
    delete [] hostArray3;
    hipFree(deviceArray1);
    hipFree(deviceArray2);
    hipFree(deviceArray3);

    return 0;
}

