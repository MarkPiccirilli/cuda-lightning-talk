
#include <hip/hip_runtime.h>
#include <omp.h>
#include <cstdlib>
#include <iostream>
#include <math.h>

#ifndef ARRAYSIZE
#define ARRAYSIZE 1000000
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE 256
#endif

#ifndef NUMBLOCKS
#define NUMBLOCKS NULL
#endif

#ifndef NUMTRIES
#define NUMTRIES 5
#endif

using std::cout;
using std::cerr;
using std:: endl;

__global__
void cudaMultiply(int *array1, int *array2, int *array3, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < size; i += stride)
        array3[i] = array1[i] * array2[i];
}

int main(int argc, char **argv) {

    #ifndef _OPENMP
    cerr << "No OpenMP support";
    return 1;
    #endif

    int arraySize = ARRAYSIZE;
    int threadArray[] = {1, 2, 4, 6, 8, 12, 16};
    int threadArraySize = sizeof(threadArray)/sizeof(threadArray[0]);

    int* hostArray1 = new int[arraySize];
    int* hostArray2 = new int[arraySize];
    int* hostArray3 = new int[arraySize];

    for(int i = 0; i < arraySize; i++) {
        hostArray1[i] = rand() % 100;
        hostArray2[i] = rand() % 100;
    }

    for(int i = 0; i < threadArraySize; i++) {
        omp_set_num_threads(threadArray[i]);

        double totalTime = 0;
        for(int j = 0; j < NUMTRIES; j++) {
            double timeStart = omp_get_wtime();
            #pragma omp parallel for
            for(int k = 0; k < arraySize; k++) {
                hostArray3[k] = hostArray1[k] * hostArray2[k];
            }
            double timeComplete = omp_get_wtime();
            totalTime = totalTime + timeComplete - timeStart;
        }
        double averageTime = totalTime / NUMTRIES;

        cout << "Time to complete operation with " << threadArray[i] << " threads(ms): " << averageTime * 1000 << '\n';
    }

    int *deviceArray1, *deviceArray2, *deviceArray3;
    hipMallocManaged(&deviceArray1, arraySize * sizeof(int));
    hipMallocManaged(&deviceArray2, arraySize * sizeof(int));
    hipMallocManaged(&deviceArray3, arraySize * sizeof(int));

    hipMemcpy(deviceArray1, hostArray1, arraySize * sizeof(int), hipMemcpyHostToDevice);

    //allocate CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int blockSize = BLOCKSIZE;
    int numBlocks = NUMBLOCKS ? NUMBLOCKS : (arraySize + blockSize - 1) / blockSize;

    cout << "Block Size: " << blockSize << endl;
    cout << "Number of Blocks: " << numBlocks << endl;

    double totalTimeGPU = 0.0;
    for(int i = 0; i < NUMTRIES; i++) {
        hipEventRecord(start, NULL);
        cudaMultiply<<<numBlocks, blockSize>>>(deviceArray1, deviceArray2, deviceArray3, arraySize);
        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop);

        float runTimeGPU = 0.0f;
        hipEventElapsedTime(&runTimeGPU, start, stop);

        totalTimeGPU = totalTimeGPU + runTimeGPU;
    }

    double averageTimeGPU = totalTimeGPU / NUMTRIES;

    cout << "Time to complete operation with the GPU using blocksize: " << blockSize << " and numBlocks: " << numBlocks << " was(ms): " << averageTimeGPU << '\n';

    // Free memory
    delete [] hostArray1;
    delete [] hostArray2;
    delete [] hostArray3;
    hipFree(deviceArray1);
    hipFree(deviceArray2);
    hipFree(deviceArray3);

    return 0;
}

